

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define N 8
#define thread_num 4
#define block_num 2


__global__ void prescan(float *g_odata, float *g_idata, int n);
void scanCPU(float *f_out, float *f_in, int i_n);

double myDiffTime(struct timeval &start, struct timeval &end)
{
double d_start, d_end;
d_start = (double)(start.tv_sec + start.tv_usec/1000000.0);
d_end = (double)(end.tv_sec + end.tv_usec/1000000.0);
return (d_end - d_start);
}

int main()
{
float a[N], c[N], g[N];
timeval start, end;

float *dev_a, *dev_g;
int size = N * sizeof(float);

double d_gpuTime, d_cpuTime;

// initialize matrices a
for (int i = 0; i < N; i++)
{
//        a[i] = (float)(rand() % 1000000) / 1000.0;
    a[i] = i+1;
    printf("a[%i] = %f\n", i, a[i]);
}
// initialize a and b matrices here
hipMalloc((void **) &dev_a, size);
hipMalloc((void **) &dev_g, size);

gettimeofday(&start, NULL);

hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

prescan<<<block_num,thread_num,2*thread_num*sizeof(float)>>>(dev_g, dev_a, N);
hipDeviceSynchronize();

hipMemcpy(g, dev_g, size, hipMemcpyDeviceToHost);

gettimeofday(&end, NULL);
d_gpuTime = myDiffTime(start, end);

gettimeofday(&start, NULL);
scanCPU(c, a, N);

gettimeofday(&end, NULL);
d_cpuTime = myDiffTime(start, end);

hipFree(dev_a); hipFree(dev_g);

for (int i = 0; i < N; i++)
{
    printf("c[%i] = %0.3f, g[%i] = %0.3f\n", i, c[i], i, g[i]);
}

printf("GPU Time for scan size %i: %f\n", N, d_gpuTime);
printf("CPU Time for scan size %i: %f\n", N, d_cpuTime);
}


__global__ void prescan(float *g_odata, float *g_idata, int n)
{
extern  __shared__  float temp[];
// allocated on invocation
int thid = threadIdx.x;
int bid = blockIdx.x;


int offset = 1;
if((bid * thread_num + thid)<n){ temp[thid]  = g_idata[bid * thread_num + thid];
}else{ temp[thid]  = 0;
} // Make the "empty" spots zeros, so it won't affect the final result.

for (int d = thread_num>>1; d > 0; d >>= 1)
    // build sum in place up the tree
{
    __syncthreads();
    if (thid < d)
    {
        int ai = offset*(2*thid+1)-1;
        int bi = offset*(2*thid+2)-1;
        temp[bi] += temp[ai];
    }
    offset *= 2;
}

if (thid == 0)
{
    temp[thread_num - 1] = 0;
}

// clear the last element
for (int d = 1; d < thread_num; d *= 2)
    // traverse down tree & build scan
{
    offset >>= 1;
    __syncthreads();
    if (thid < d)
    {
        int ai = offset*(2*thid+1)-1;
        int bi = offset*(2*thid+2)-1;
        float t = temp[ai];
        temp[ai]  = temp[ bi];
        temp[bi] += t;
    }
}
__syncthreads();

g_odata[bid * thread_num + thid] = temp[thid];
}

void scanCPU(float *f_out, float *f_in, int i_n)
{
f_out[0] = 0;
for (int i = 1; i < i_n; i++)
    f_out[i] = f_out[i-1] + f_in[i-1];

}