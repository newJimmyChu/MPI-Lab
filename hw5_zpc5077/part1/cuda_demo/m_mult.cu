
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define N 64

__global__ void matrixMult (int *a, int *b, int *c, int width);
void matrixMultCPU (int a[N][N], int b[N][N], int c[N][N], int width);


double myDiffTime(struct timeval &start, struct timeval &end)
{
	double d_start, d_end;
	d_start = (double)(start.tv_sec + start.tv_usec/1000000.0);
	d_end = (double)(end.tv_sec + end.tv_usec/1000000.0);
	return (d_end - d_start);
}

int main() 
{
	int a[N][N], b[N][N], c[N][N], g[N][N];
	int *dev_a, *dev_b, *dev_c;
	timeval start, end;

	// initialize matrices a and b with appropriate values
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			a[i][j] = i*N + j;
			b[i][j] = i + j;
		}
	}

	int size = N * N * sizeof(int);

	hipMalloc((void **) &dev_a, size);
	hipMalloc((void **) &dev_b, size);
	hipMalloc((void **) &dev_c, size);

	gettimeofday(&start, NULL);	

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

	dim3 dimGrid(1, 1);
	dim3 dimBlock(N, N);

	matrixMult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, N);
	hipDeviceSynchronize();
	hipMemcpy(g, dev_c, size, hipMemcpyDeviceToHost);

	gettimeofday(&end, NULL);
	printf("GPU Time for %i additions: %f\n", N, myDiffTime(start, end));

	gettimeofday(&start, NULL);
	matrixMultCPU(a, b, c, N);
	
	gettimeofday(&end, NULL);
	printf("CPU Time for %i additions: %f\n", N, myDiffTime(start, end));

	hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);

	// print verification
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			if (c[i][j] != g[i][j])
			{
				printf("Results do not match! %i, %i, c=%i, g=%i\n", i, j, c[i][j], g[i][j]);
				exit(1);
			}
		}
	}


}

__global__ void matrixMult (int *a, int *b, int *c, int width) 
{
	int k, sum = 0;
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	if(col < width && row < width) 
	{
		for (k = 0; k < width; k++)
			sum += a[row * width + k] * b[k * width + col];
		c[row * width + col] = sum;
	}
}

void matrixMultCPU (int a[N][N], int b[N][N], int c[N][N], int width) 
{
	for (int i = 0; i < width; i++) 
	{
		for (int j = 0; j < width; j++) 
		{
			int sum = 0;
			for (int k = 0; k < width; k++) 
			{
				int m = a[i][k];
				int n = b[k][j];
				sum += m * n;
			}
		c[i][j] = sum;
		}
	}
}
