
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define N 1024

#define NUM_BANKS 16 
#define LOG_NUM_BANKS 4 
#ifdef ZERO_BANK_CONFLICTS 
	#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS)) 
#else 
	#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS) 
#endif

__global__ void scan(float *g_odata, float *g_idata, int n);
__global__ void prescan(float *g_odata, float *g_idata, int n);
void scanCPU(float *f_out, float *f_in, int i_n);

double myDiffTime(struct timeval &start, struct timeval &end)
{
	double d_start, d_end;
	d_start = (double)(start.tv_sec + start.tv_usec/1000000.0);
	d_end = (double)(end.tv_sec + end.tv_usec/1000000.0);
	return (d_end - d_start);
}

int main() 
{
	float a[N], c[N], g[N];
	timeval start, end;

	float *dev_a, *dev_g;
	int size = N * sizeof(float);

	double d_gpuTime, d_cpuTime;

	// initialize matrices a 
	for (int i = 0; i < N; i++)
	{
		a[i] = (float)(rand() % 1000000) / 1000.0;
		//printf("a[%i] = %f\n", i, a[i]);
	}
	// initialize a and b matrices here
	hipMalloc((void **) &dev_a, size);
	hipMalloc((void **) &dev_g, size);

	gettimeofday(&start, NULL);

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

	prescan<<<1,N,2*N*sizeof(float)>>>(dev_g, dev_a, N);
	hipDeviceSynchronize();

	hipMemcpy(g, dev_g, size, hipMemcpyDeviceToHost);

	gettimeofday(&end, NULL);
	d_gpuTime = myDiffTime(start, end);

	gettimeofday(&start, NULL);
	scanCPU(c, a, N);
	
	gettimeofday(&end, NULL);
	d_cpuTime = myDiffTime(start, end);
	

	hipFree(dev_a); hipFree(dev_g);

	for (int i = 0; i < N; i++)
	{
		printf("c[%i] = %0.3f, g[%i] = %0.3f\n", i, c[i], i, g[i]);
		//if (c[i] != g[i])
		//{
		//	printf("Results do not match! c[%i]=%f, g[%i]=%f\n", i, c[i], i, g[i]);
		//	break;
		//}
	}

	printf("GPU Time for scan size %i: %f\n", N, d_gpuTime);
	printf("CPU Time for scan size %i: %f\n", N, d_cpuTime);
}

__global__ void prescan(float *g_odata, float *g_idata, int n) 
{ 
	extern  __shared__  float temp[];
	// allocated on invocation 
	int thid = threadIdx.x; 
	int offset = 1; 

	//A
	int ai = thid; 
	int bi = thid + (n/2); 
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai); 
	int bankOffsetB = CONFLICT_FREE_OFFSET(ai); 
	temp[ai + bankOffsetA] = g_idata[ai];  
	temp[bi + bankOffsetB] = g_idata[bi];  

 
	for (int d = n>>1; d > 0; d >>= 1) 
	// build sum in place up the tree 
    	{ 
        	__syncthreads(); 
		if (thid < d)    
        	{ 
			//B
			int ai = offset*(2*thid+1)-1; 
			int bi = offset*(2*thid+2)-1; 
			ai += CONFLICT_FREE_OFFSET(ai); 
			bi += CONFLICT_FREE_OFFSET(bi);  

		    	temp[bi] += temp[ai];         
  		} 
        	offset *= 2; 
    	} 

	//C
	if (thid==0) 
	{ 
		int z = CONFLICT_FREE_OFFSET(n - 1);
		temp[n - 1 + z] = 0; 
	}

	// clear the last element 
	for (int d = 1; d < n; d *= 2) 
	// traverse down tree & build scan 
    	{ 
        	offset >>= 1; 
        	__syncthreads(); 
		if (thid < d) 
        	{ 
			//D
			int ai = offset*(2*thid+1)-1; 
			int bi = offset*(2*thid+2)-1; 
			ai += CONFLICT_FREE_OFFSET(ai); 
			bi += CONFLICT_FREE_OFFSET(bi); 

			float t   = temp[ai]; 
            		temp[ai]  = temp[bi]; 
            		temp[bi] += t; 
        	} 
    	} 
    	__syncthreads(); 

	//E
	g_odata[ai] = temp[ai + bankOffsetA];  
	g_odata[bi] = temp[bi + bankOffsetB];   
}
 
void scanCPU(float *f_out, float *f_in, int i_n)
{
	f_out[0] = 0;
	for (int i = 1; i < i_n; i++)
		f_out[i] = f_out[i-1] + f_in[i-1];

}
